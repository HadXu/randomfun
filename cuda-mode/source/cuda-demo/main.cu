
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#define DSIZE 8
#define SCF 0.5f
#define nTPB 256
__global__ void half_scale_kernel(float *din, float *dout, int dsize){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx < dsize){
    half scf = __float2half(SCF);
    half kin = __float2half(din[idx]);
    half kout;
    

#if __CUDA_ARCH__ >= 530
    printf("__CUDA_ARCH__ >= 530\n");
    kout = __hmul(kin, scf);
#else
    kout = __float2half(__half2float(kin)*__half2float(scf));
#endif
    dout[idx] = __half2float(kout);
    }
}

int main(){

  float *hin, *hout, *din, *dout;
  hin  = (float *)malloc(DSIZE*sizeof(float));
  hout = (float *)malloc(DSIZE*sizeof(float));
  for (int i = 0; i < DSIZE; i++) hin[i] = i;
  hipMalloc(&din,  DSIZE*sizeof(float));
  hipMalloc(&dout, DSIZE*sizeof(float));
  hipMemcpy(din, hin, DSIZE*sizeof(float), hipMemcpyHostToDevice);
  half_scale_kernel<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(din, dout, DSIZE);
  hipMemcpy(hout, dout, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < DSIZE; i++) printf("%f\n", hout[i]);
  return 0;
}